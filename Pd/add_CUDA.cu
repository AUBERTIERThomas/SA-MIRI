// add_CUDA.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

#define T 4

__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
	printf("GPU : computed %d + %d = %d\n", *a, *b, *c);
}

int main(void)
{
	int a,b,c;
	int *d_a,*d_b,*d_c;
	int size = sizeof(int);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	a = 2;
	b = 7;
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	add<<<1,T>>>(d_a,d_b,d_c);
	
	hipMemcpy(d_c, &c, size, hipMemcpyDeviceToHost);
	
	printf("CPU : received %d + %d = %d\n", a, b, c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();
	
	return 0;
}
