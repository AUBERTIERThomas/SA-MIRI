#include "hip/hip_runtime.h"
// add_CUDA.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4194304
#define TPB 

__global__ void add(int *a, int *b, int *c)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < n) c[index] = a[index] + b[index];
}

int main(void)
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	int size = N*sizeof(int);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	a = (int*)malloc(size); random_ints(a,N);
	b = (int*)malloc(size); random_ints(b,N);
	c = (int*)malloc(size);
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	add<<<N/TPB,TPB>>>(d_a,d_b,d_c);
	
	hipMemcpy(d_c, &c, size, hipMemcpyDeviceToHost);
	
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}
